#include "hip/hip_runtime.h"
#include <cuda/Parallel_functions.h> 

__global__ void skipValue_kernal(int *S, float *R, int *B, float *p);

void parallel_generateSkipValue(int *S, float *R, int B, float p) {
	int *d_S, *d_B, *h_B;
	float *d_R, *h_p, *d_p;
	
	// allocate:
	h_B = new int[1];
	h_p = new float[1];
	hipMalloc((void**) &d_S, B * sizeof(int));
	hipMalloc((void**) &d_B, sizeof(int));
	hipMalloc((void**) &d_R, B * sizeof(float));
	hipMalloc((void**) &d_p, sizeof(float));
	
	// copy:
	h_B[0] = B;
	h_p[0] = p;
	hipMemcpy(d_B, h_B, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_p, h_p, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_R, R, B * sizeof(float), hipMemcpyHostToDevice);
	
	// run kernel:
	skipValue_kernal<<<pow(2, 16)-1, pow(2, 10)>>>(d_S, d_R, d_B, d_p);
	
	// copy:
	hipMemcpy(S, d_S, B * sizeof(float), hipMemcpyDeviceToHost);
	
	// free:
	delete h_B;
	delete h_p;
	hipFree(d_S);
	hipFree(d_B);
	hipFree(d_R);
	hipFree(d_p);
}
