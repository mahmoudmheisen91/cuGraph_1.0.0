#include "hip/hip_runtime.h"
#include <cuda/Parallel_functions.h> 

__global__ void skipValue_kernal(float *S, float *R, int B, float p) {

	int k;
	float theta, logp;
    logp = log10f(1 - p);
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	
    while (tid < B) {
		theta = log10f(R[tid]) / logp;
		k = max(0, (int)ceil(theta) );
		
		S[tid] = k;
		tid += blockDim.x * gridDim.x;
	}
}

__global__ void skipValuePre_kernal(float *S, float *R, int *B, float *p, int *m, float *F) {
	
	int k;
	float theta;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
		
	while (tid < *B) {
		
		int j = 0;
		while (j <= *m) {
			if(F[j] > R[tid]) {
				k = j;
				break;       // to break from while loop;
			}
			else
				j++;
		}
		
		if(j == *m + 1) {
			theta = log10f(R[tid]) / log10f(1 - *p);
			k = max(0, (int)ceil(theta) );
		}
		
		S[tid] = k;
		tid += blockDim.x * gridDim.x;
	}
}
