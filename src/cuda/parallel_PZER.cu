#include "hip/hip_runtime.h"
#include <cuda/Parallel_functions.h>
#include <iostream>

__global__ void random_number_generator_kernal(int masterSeed, int size, float *PRNG);
__global__ void skipValue_kernal(float *S, float *R, int B, float p);
__global__ void skipValuePre_kernal(float *S, float *R, int B, float p, int m, float *F);
__global__ void addEdges_kernal(bool *content, float *S, int V, int B, int *L, int last);
__global__ void generatePredicateList_kernel(float *PL, int *T, float *R, int B, int i, float p);
__global__ void compact_kernel(int *T, float *S, float *PL, int *SC, int B);
__global__ void addEdges_kernel2(bool *content, float *SC, int V, int B);

void initDevice(void) {
    hipFree(0);
}

void parallel_PZER(bool *content, float p, int lambda, int V, int E) {
    // declerations:
    bool *d_content;
    float *d_R, *d_S;
    int *d_L, *h_L;

    int B, L = 0;
    int seed = time(0)-1000000000;
    double segma = sqrt(p * (1 - p) * E);

    if((int)(p * E + lambda * segma) < 1000000)
        B = (int)(p * E + lambda * segma);
    else
        B = 1000000;

    // allocation:
    h_L = new int[1];
    hipMalloc((void**) &d_content, V * V * sizeof(bool)); 	// 100 MB
    hipMalloc((void**) &d_R, B * sizeof(float)); 			// 4 MB
    hipMalloc((void**) &d_S, B * sizeof(float));			// 4 MB
    hipMalloc((void**) &d_L, sizeof(int));
    thrust::device_ptr<float> d = thrust::device_pointer_cast(d_S);

    // copy:
    hipMemcpy(d_content, content, V * V * sizeof(bool), hipMemcpyHostToDevice);

    // run kernals:
    while(L < E) {
        random_number_generator_kernal<<<8, pow(2, 10)>>>(seed, B, d_R);
        skipValue_kernal<<<32, pow(2, 10)>>>(d_S, d_R, B, p);
        thrust::inclusive_scan(d, d+B, d);
        addEdges_kernal<<<32, pow(2, 10)>>>(d_content, raw_pointer_cast(&d[0]), V, B, d_L, L);

        hipMemcpy(h_L, d_L, sizeof(float), hipMemcpyDeviceToHost);
        L = h_L[0];

        //std::cout << L << " " << last << std::endl;
    }

    hipMemcpy(content, d_content, sizeof(bool) * V * V, hipMemcpyDeviceToHost);

    // free:
    delete h_L;
    hipFree(d_content);
    hipFree(d_R);
    hipFree(d_S);
    hipFree(d_L);
}

void parallel_PPreZER(bool *content, float p, int lambda, int m, int V, int E) {
    // declerations:
    bool *d_content;
    float *d_R, *d_S, *h_F, *d_F;
    int *d_L, *h_L;

    int B, L = 0;
    int seed = time(0)-1000000000;
    double segma = sqrt(p * (1 - p) * E);

    if((int)(p * E + lambda * segma) < 1000000)
        B = (int)(p * E + lambda * segma);
    else
        B = 1000000;

    h_F = new float[m+1];
    for(int i = 0; i <= m; i++) {
        h_F[i] = 1 - pow(1-p, i+1);
    }

    // allocation:
    h_L = new int[1];
    hipMalloc((void**) &d_L, sizeof(int));
    hipMalloc((void**) &d_F, (m+1) * sizeof(float));
    hipMalloc((void**) &d_content, V * V * sizeof(bool)); 	// 100 MB
    hipMalloc((void**) &d_R, B * sizeof(float)); 			// 4 MB
    hipMalloc((void**) &d_S, B * sizeof(float));			// 4 MB
    thrust::device_ptr<float> d = thrust::device_pointer_cast(d_S);

    // copy:
    hipMemcpy(d_F, h_F, (m+1) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_content, content, V * V * sizeof(bool), hipMemcpyHostToDevice);

    // run kernals:
    while(L < E) {
        random_number_generator_kernal<<<1, pow(2, 10)>>>(seed, B, d_R);
        skipValuePre_kernal<<<32, pow(2, 10)>>>(d_S, d_R, B, p, m, d_F);
        thrust::inclusive_scan(d, d+B, d);
        addEdges_kernal<<<32, pow(2, 10)>>>(d_content, raw_pointer_cast(&d[0]), V, B, d_L, L);

        hipMemcpy(h_L, d_L, sizeof(float), hipMemcpyDeviceToHost);
        L = h_L[0];

        //std::cout << h_last[0] << std::endl;
    }

    hipMemcpy(content, d_content, sizeof(bool) * V * V, hipMemcpyDeviceToHost);

    // free:
    delete h_L;
    delete h_F;
    hipFree(d_L);
    hipFree(d_content);
    hipFree(d_R);
    hipFree(d_S);
    hipFree(d_F);
}

void parallel_PER(bool *content, float p, int V, int E) {

    // declerations:
    bool *d_content;
    float *d_R, *d_PL;
    int *d_T, *d_SC;

    int B = 1000000;
    int seed = time(0)-1000000000;
    int iter = E / B;

    // allocation:
    hipMalloc((void**) &d_content, V * V * sizeof(bool)); 	// 100 MB
    hipMalloc((void**) &d_R, B * sizeof(float)); 			// 8 MB
    hipMalloc((void**) &d_PL, B * sizeof(float)); 			// 8 MB
    hipMalloc((void**) &d_T, B * sizeof(int)); 			// 8 MB
    hipMalloc((void**) &d_SC, B * sizeof(int)); 			// 8 MB

    // copy:
    hipMemcpy(d_content, content, V * V * sizeof(bool), hipMemcpyHostToDevice);

    // run kernals:
    for(int i = 0; i < iter; i++) {
        random_number_generator_kernal	<<<1 , pow(2, 10)>>> (seed, B, d_R);
        generatePredicateList_kernel	<<<32, pow(2, 10)>>> (d_PL, d_T, d_R, B, i, p);
        //thrust::inclusive_scan(d, d+B, d);
        //compact_kernel					<<<32, pow(2, 10)>>> (d_T, raw_pointer_cast(&d[0]), d_PL, d_SC, B);
        addEdges_kernel2				<<<32, pow(2, 10)>>> (d_content, d_PL, V, B);
    }

    hipMemcpy(content, d_content, sizeof(bool) * V * V, hipMemcpyDeviceToHost);

    // free:
    hipFree(d_content);
    hipFree(d_R);
    hipFree(d_SC);
    hipFree(d_T);
    hipFree(d_PL);
}

__global__ void generatePredicateList_kernel(float *PL, int *T, float *R, int B, int i, float p) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while(tid < B) {

        T[tid] = tid + i * B;

        if (R[tid] < p)
            PL[tid] = tid + i * B;
        else
            PL[tid] = -1;

        tid += blockDim.x * gridDim.x;
    }
}

__global__ void compact_kernel(int *T, float *S, float *PL, int *SC, int B) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while(tid < B) {

        if((int)PL[tid] == 1) {
            float j = S[tid];
            SC[(int)j] = T[tid];
        }

        tid += blockDim.x * gridDim.x;
    }
}

