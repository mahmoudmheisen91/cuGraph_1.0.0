#include "hip/hip_runtime.h"
#include <cuda/Parallel_functions.h> 
#include <iostream>

__global__ void random_number_generator_kernal(int *masterSeed, int *size, float *PRNG);
__global__ void skipValue_kernal(float *S, float *R, int *B, float *p);
__global__ void skipValuePre_kernal(float *S, float *R, int *B, float *p, int *m, float *F);
__global__ void addEdges_kernal(bool *content, float *S, int *V, int *B, float *L, float *last);

void initDevice(void) {
	hipFree(0);
}

void parallel_PZER(bool *content, float p, int lambda, int V, int E) {
	// declerations:
	bool *d_content;
	float *d_R, *d_S, *d_p, *h_p, *d_odata, *d_L, *h_L, *h_last, *d_last, *h_S;
	int *d_seed, *h_seed, *d_B, *h_B, *d_V, *h_V;
	
	int B, L = 0;
    int seed = time(0)-1000000000;
    double segma = sqrt(p * (1 - p) * E);

    if((int)(p * E + lambda * segma) < 2000000)
        B = (int)(p * E + lambda * segma);
    else
        B = 2000000;

	// allocation:
	h_p = new float[1];
	h_seed = new int[1];
	h_V = new int[1];
	h_B = new int[1];
	h_L = new float[1];
	h_last = new float[1];
	h_S = new float[B];
	hipMalloc((void**) &d_p, sizeof(float));
	hipMalloc((void**) &d_seed, sizeof(int));
	hipMalloc((void**) &d_V, sizeof(int));
	hipMalloc((void**) &d_B, sizeof(int));
	hipMalloc((void**) &d_L, sizeof(float));
	hipMalloc((void**) &d_last, sizeof(int));
	hipMalloc((void**) &d_content, V * V * sizeof(bool)); 	// 100 MB
	hipMalloc((void**) &d_R, B * sizeof(float)); 			// 8 MB 
	hipMalloc((void**) &d_S, B * sizeof(float));			// 8 MB
	hipMalloc((void**) &d_odata, B * sizeof(float));		// 8 MB
	
	// fill:
	h_p[0] = p;
	h_seed[0] = seed;
	h_V[0] = V;
	h_B[0] = B;
	
	// copy:
	hipMemcpy(d_p, h_p, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_seed, h_seed, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_V, h_V, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_L, h_L, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_content, content, V * V * sizeof(bool), hipMemcpyHostToDevice);
	
	srand(time(0));
	h_last[0] = 0;
	
	// run kernals:
	while(L < E) {
		hipMemcpy(d_last, h_last, sizeof(float), hipMemcpyHostToDevice);
		
		random_number_generator_kernal<<<1, pow(2, 10)>>>(d_seed, d_B, d_R);
		skipValue_kernal<<<32, pow(2, 10)>>>(d_S, d_R, d_B, d_p);	
		preallocBlockSums(B);
		prescanArray(d_odata, d_S, B);
		addEdges_kernal<<<32, pow(2, 10)>>>(d_content, d_odata, d_V, d_B, d_L, d_last);
		
		// copy:
		hipMemcpy(h_L, d_L, sizeof(float), hipMemcpyDeviceToHost);
		L = (int)h_L[0];
		
		hipMemcpy(h_S, d_odata, B*sizeof(float), hipMemcpyDeviceToHost);
		h_last[0] = h_S[B-1];
		//std::cout << h_last[0] << std::endl;
	}
	
	hipMemcpy(content, d_content, sizeof(bool) * V * V, hipMemcpyDeviceToHost);
	
	// free:
	delete h_p;
	delete h_seed;
	delete h_B;
	delete h_V;
	delete h_L;
	delete h_last;
	delete h_S;
	hipFree(d_p);
	hipFree(d_seed);
	hipFree(d_B);
	hipFree(d_V);
	hipFree(d_L);
	hipFree(d_content);
	hipFree(d_R);
	hipFree(d_S);	
	deallocBlockSums();
    hipFree(d_odata);
    hipFree(d_last);
}

void parallel_PPreZER(bool *content, float p, int lambda, int m, int V, int E) {
	// declerations:
	bool *d_content;
	float *d_R, *d_S, *d_p, *h_p, *d_odata, *d_L, *h_L, *h_last, *d_last, *h_S, *h_F, *d_F;
	int *d_seed, *h_seed, *d_B, *h_B, *d_V, *h_V, *d_m, *h_m;
	
	int B, L = 0;
    int seed = time(0)-1000000000;
    double segma = sqrt(p * (1 - p) * E);

    if((int)(p * E + lambda * segma) < 2000000)
        B = (int)(p * E + lambda * segma);
    else
        B = 2000000;

	float *F = new float[m+1];
    for(int i = 0; i <= m; i++) {
        F[i] = 1 - pow(1-p, i+1);
    }
        
	// allocation:
	h_p = new float[1];
	h_seed = new int[1];
	h_V = new int[1];
	h_B = new int[1];
	h_L = new float[1];
	h_last = new float[1];
	h_S = new float[B];
	h_F = new float[m+1];
	h_m = new int[1];
	hipMalloc((void**) &d_p, sizeof(float));
	hipMalloc((void**) &d_seed, sizeof(int));
	hipMalloc((void**) &d_V, sizeof(int));
	hipMalloc((void**) &d_B, sizeof(int));
	hipMalloc((void**) &d_L, sizeof(float));
	hipMalloc((void**) &d_last, sizeof(int));
	hipMalloc((void**) &d_m, sizeof(int));
	hipMalloc((void**) &d_F, (m+1) * sizeof(float));
	hipMalloc((void**) &d_content, V * V * sizeof(bool)); 	// 100 MB
	hipMalloc((void**) &d_R, B * sizeof(float)); 			// 8 MB 
	hipMalloc((void**) &d_S, B * sizeof(float));			// 8 MB
	hipMalloc((void**) &d_odata, B * sizeof(float));		// 8 MB
	
	// fill:
	h_p[0] = p;
	h_seed[0] = seed;
	h_V[0] = V;
	h_B[0] = B;
	h_m[0] = m;
	
	// copy:
	hipMemcpy(d_p, h_p, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_seed, h_seed, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_V, h_V, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_L, h_L, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_F, h_F, (m+1) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_m, h_m, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_content, content, V * V * sizeof(bool), hipMemcpyHostToDevice);
	
	srand(time(0));
	h_last[0] = 0;
	
	// run kernals:
	while(L < E) {
		hipMemcpy(d_last, h_last, sizeof(float), hipMemcpyHostToDevice);
		
		random_number_generator_kernal<<<1, pow(2, 10)>>>(d_seed, d_B, d_R);
		skipValuePre_kernal<<<32, pow(2, 10)>>>(d_S, d_R, d_B, d_p, d_m, d_F);	
		preallocBlockSums(B);
		prescanArray(d_odata, d_S, B);
		addEdges_kernal<<<32, pow(2, 10)>>>(d_content, d_odata, d_V, d_B, d_L, d_last);
		
		// copy:
		hipMemcpy(h_L, d_L, sizeof(float), hipMemcpyDeviceToHost);
		L = (int)h_L[0];
		
		hipMemcpy(h_S, d_odata, B*sizeof(float), hipMemcpyDeviceToHost);
		h_last[0] = h_S[B-1];
		//std::cout << h_last[0] << std::endl;
	}
	
	hipMemcpy(content, d_content, sizeof(bool) * V * V, hipMemcpyDeviceToHost);
	
	// free:
	delete h_p;
	delete h_seed;
	delete h_B;
	delete h_V;
	delete h_L;
	delete h_last;
	delete h_S;
	delete h_m;
	delete h_F;
	hipFree(d_p);
	hipFree(d_seed);
	hipFree(d_B);
	hipFree(d_V);
	hipFree(d_L);
	hipFree(d_content);
	hipFree(d_R);
	hipFree(d_S);	
	deallocBlockSums();
    hipFree(d_odata);
    hipFree(d_last);
    hipFree(d_m);
    hipFree(h_F);
}

void parallel_PER(bool *content, float p, int V, int E) {

}










