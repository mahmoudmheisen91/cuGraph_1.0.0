#include "hip/hip_runtime.h"
#include <cuda/Parallel_functions.h> 

__global__ void addEdges_kernal(bool *content, float *S, int *V, int *B, float *L, float *last) {
	
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int v1, v2;
	
	while (tid < *B) {
		S[tid] += *last;
		v1 = (int)S[tid] / *V;
		v2 = (int)S[tid] % *V;
		content[v1 * *V + v2] = 1;
		content[v2 * *V + v1] = 1;
		
		if(tid == *B-1) L[0] = S[tid];
		
		tid += blockDim.x * gridDim.x;
	}	
}
