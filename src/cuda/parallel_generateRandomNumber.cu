#include "hip/hip_runtime.h"
#include <cuda/Parallel_functions.h> 

__global__ void random_number_generator_kernal(int *masterSeed, int *size, float *PRNG);

void parallel_generateRandomNumber(float *R, int B, int seed) {

    // declare host and device variable:
	int *host_masterSeed, *host_size, *dev_masterSeed, *dev_size;
	float *dev_PRNG;
	
	dim3 grid(1);
	dim3 block(pow(2, 10));

	// allocate host memory:
	host_masterSeed = new int[1];
	host_size = new int[1];
	
	// allocate device memory:
	hipMalloc((void**)&dev_masterSeed, sizeof(int));
	hipMalloc((void**)&dev_size, sizeof(int));
	hipMalloc((void**)&dev_PRNG, B * sizeof(float));
	
	// fill 
	host_masterSeed[0] = seed; // between 1 and m-1
	host_size[0] = B;
	
	// copy host vars to device vars:
	hipMemcpy(dev_masterSeed, host_masterSeed, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_size, host_size, sizeof(int), hipMemcpyHostToDevice);

	// start Device kernal:
	random_number_generator_kernal<<<grid, block>>>(dev_masterSeed, dev_size, dev_PRNG);
	
	// copy device vars to host vars:
	hipMemcpy(R, dev_PRNG, B * sizeof(float), hipMemcpyDeviceToHost);
	
	// free host memory:
	delete host_masterSeed;
	delete host_size;
	
	// free device memory:
	hipFree(dev_masterSeed);
	hipFree(dev_size);
	hipFree(dev_PRNG);
}
