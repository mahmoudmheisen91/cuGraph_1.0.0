#include "hip/hip_runtime.h"
/*
 * scan_kernels.cu
 *
 *  Created: 2015-06-18, Modified: 2015-07-26
 *
 */

// Headers includes:
#include <cuda/kernels.cuh>
#include <cuda/Cuda_Prototypes_Macros.h>

/** Single warp scan algorithm.
 * device function that scan a single warp of threads
 */
__device__ int single_warp_scan(int *data, /* in\out */ 
								int idx)  /* in */ 
{
	const unsigned int tid = threadIdx.x;					 // thread id in a block: (0 to 1023)
	const unsigned int lane = tid & 31; 					 // index of thread in warp (0..31):
	
	// Unroll the loop:
	if ( lane >= 1)  
		data[idx] = data[idx] + data[idx - 1];
		
	if ( lane >= 2)  
		data[idx] = data[idx] + data[idx - 2];
		
	if ( lane >= 4)  
		data[idx] = data[idx] + data[idx - 4];
		
	if ( lane >= 8)  
		data[idx] = data[idx] + data[idx - 8];
		
	if ( lane >= 16) 
		data[idx] = data[idx] + data[idx - 16];
	
	return data[idx];
}

/** Single block scan algorithm.
 * device function that scan a single block of threads
 * internally depend on single_warp_scan
 */								
__device__ int single_block_scan(int *data, /* in\out */ 
								 int idx)   /* in */ 
{	
	const unsigned int tid = threadIdx.x;  	// thread id in a block: (0 to 1023)
	const unsigned int bid = blockIdx.x;  	// block id in a grid: (0 to 1023)
	const unsigned int bdim = blockDim.x;  	// block size: 1024
	const unsigned int lane = tid & 31;  	// thread id in a warp: (0 to 31)
	const unsigned int warpid = tid >> 5; 	// warp id in a block: (0 to 31)
	
	// Step 1: Single warp scan:
	int val = single_warp_scan(data, idx);
	__syncthreads();
		
	// Step 2: Collect partial results per warp:
	if( lane == 31 ) { 
		data[warpid + (bid*bdim) ] = data[idx]; // last thread in each warp
	}
	__syncthreads();
	
	// Step 3: Scan partail results:
	if( warpid == 0) {
		single_warp_scan(data, idx);
	}
	__syncthreads();

	// Step 4: Accumulate results from Steps 1 and 3:
	if (warpid > 0) {
		val = data[(warpid - 1) + (bid*bdim)] + val;
	}
	__syncthreads();
	
	// Step 5: Write and return the final result:
	data[idx] = val;
	__syncthreads();
	
	return val ;
}

/** First kernel of global scan algorithm.
 * scan data array as blocks
 * store partail results in block_results
 */										
__global__ void global_scan_kernel_1(int *data, 			/* in\out */ 
									 int *block_results)	/* out */ 
{
	const unsigned int tid = threadIdx.x;				// thread id in a block: (0 to 1023)
	const unsigned int bid = blockIdx.x;				// block id in a grid: (0 to 1023)
	const unsigned int gid = tid + bid * blockDim.x;  	// global id of the thread
	
	// step 1: block scan:
	int val = single_block_scan(data, gid);
	__syncthreads();
	
	// step 2: store partial result from each block:
	if (tid == 1023) {
		block_results[bid] = data[gid];
	}
	__syncthreads();
}

/** Second kernel of global scan algorithm.
 * scan block_results array
 */									
__global__ void global_scan_kernel_2(int *block_results)	/* in\out */ 
{
	const unsigned int tid = threadIdx.x;				// thread id in a block: (0 to 1023)
	
	// step 3: block scan of block_results:
	single_block_scan(block_results, tid);
}

/** Third kernel of global scan algorithm.
 * accumalte data from scanned block_results
 * store partail results in block_results
 */	
__global__ void global_scan_kernel_3(int *data, 			/* in\out */ 
									 int *block_results)	/* in */ 
{
	const unsigned int tid = threadIdx.x;				// thread id in a block: (0 to 1023)
	const unsigned int bid = blockIdx.x;				// block id in a grid: (0 to 1023)
	const unsigned int gid = tid + bid * blockDim.x;  	// global id of the thread
	
	// Step 4: Each thread of block i adds element i from Step 3 to its output element from Step 1:
	if (bid > 0) {
		int val = block_results[bid - 1];
		data[gid] += val;
	}
}






