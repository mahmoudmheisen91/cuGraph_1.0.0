#include <cuda/Parallel_functions.h> 

void parallel_scan(int num_elements, float* h_data) {   
    // allocate device memory input and output arrays
    float* d_idata = NULL;
    float* d_odata = NULL;
    hipMalloc( (void**) &d_idata, sizeof(float) * num_elements);
    hipMalloc( (void**) &d_odata, sizeof(float) * num_elements);
    
    // copy host memory to device input array
    hipMemcpy( d_idata, h_data, sizeof(float) * num_elements, hipMemcpyHostToDevice);
    
    // run 
    preallocBlockSums(num_elements);
    prescanArray(d_odata, d_idata, num_elements);
    
    // copy device memory to host input array
    hipMemcpy(h_data, d_odata, sizeof(float) * num_elements, hipMemcpyDeviceToHost);
    
    deallocBlockSums();
    hipFree(d_odata);
    hipFree(d_idata);
}

