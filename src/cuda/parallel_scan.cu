#include <cuda/Parallel_functions.h> 

void parallel_scan(float* S, int B) {   
    // allocate device memory input and output arrays
    float* d_idata = NULL;
    float* d_odata = NULL;
    hipMalloc( (void**) &d_idata, sizeof(float) * B);
    hipMalloc( (void**) &d_odata, sizeof(float) * B);
    
    // copy host memory to device input array
    hipMemcpy( d_idata, S, sizeof(float) * B, hipMemcpyHostToDevice);
    
    // run 
    preallocBlockSums(B);
    prescanArray(d_odata, d_idata, B);
    
    // copy device memory to host input array
    hipMemcpy(S, d_odata, sizeof(float) * B, hipMemcpyDeviceToHost);
    
    deallocBlockSums();
    hipFree(d_odata);
    hipFree(d_idata);
}

