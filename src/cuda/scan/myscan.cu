#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdio>

#include "scan_kernels.cuh"

using namespace std;

int main() {

	// params:
	int size = 1024*1024;
	
	// allocate host:
	int *data_host = NULL;
	data_host = new int[size];
	
	// allocate device:
	int *data_device = NULL;
	hipMalloc((void**) &data_device, size * sizeof(int));
	
	int *block_results = NULL;
	hipMalloc((void**) &block_results, 1024 * sizeof(int));
	
	// fill host:
	for(int i = 0; i < size; i++) {
		data_host[i] = 1;
	}
	
	// copy host to device:
	hipMemcpy(data_device, data_host, size * sizeof(int), hipMemcpyHostToDevice);
	
	// kernel:
	
	float time;
    hipEvent_t start, stop; 

    hipEventCreate(&start); 
    hipEventCreate(&stop);
    hipEventRecord(start, 0); 

	global_scan_kernel_1 <<<1024, 1024>>> (data_device, block_results);
	global_scan_kernel_2 <<<1, 1024>>> (block_results);
	global_scan_kernel_3 <<<1024, 1024>>> (data_device, block_results);
	
    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop);  
    hipEventElapsedTime(&time, start, stop); 
    hipEventDestroy(start); 
    hipEventDestroy(stop);
	
	// copy device to host:
	hipMemcpy(data_host, data_device, size * sizeof(int), hipMemcpyDeviceToHost);
	
	// print:
	cout << "time = " << time << endl;
	for(int i = size-1; i < size; i++) {
		cout << data_host[i] << " ";
	}
	cout << endl;
	
	// free:
	delete data_host;
	hipFree(data_device);
	
	// end:
	return 0;
}












