#include "hip/hip_runtime.h"
#include <cuda/Parallel_functions.h> 

__global__ void addEdges_kernal(bool *content, float *S, int *V, int *B);

void parallel_addEdges(bool *content, float *S, int numberOfVertices, int B) {
	int *d_B, *h_B, *d_V, *h_V;
	float *d_S;
	bool *d_content;
	
	// allocate:
	h_B = new int[1];
	h_V = new int[1];
	hipMalloc((void**) &d_B, sizeof(int));
	hipMalloc((void**) &d_V, sizeof(int));
	
	hipMalloc((void**) &d_S, B * sizeof(float));
	hipMalloc((void**) &d_content, numberOfVertices * numberOfVertices * sizeof(bool));
	
	// copy:
	h_B[0] = B;
	h_V[0] = numberOfVertices;
	hipMemcpy(d_B, h_B, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_V, h_V, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_S, S, B * sizeof(float), hipMemcpyHostToDevice);
	
	// run kernel:
	addEdges_kernal<<<pow(2, 16)-1, pow(2, 10)>>>(d_content, d_S, d_V, d_B);
	
	// copy:
	hipMemcpy(content, d_content, numberOfVertices * numberOfVertices * sizeof(bool), hipMemcpyDeviceToHost);
	
	// free:
	delete h_B;
	delete h_V;
	hipFree(d_S);
	hipFree(d_B);
	hipFree(d_content);
	hipFree(d_V);
}
