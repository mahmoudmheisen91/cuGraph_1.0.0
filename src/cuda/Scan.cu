#include "hip/hip_runtime.h"
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

#include <cuda/Scan.h> 

template <bool isNP2>
__device__ void loadSharedChunkFromMem(float *s_data,
                                       const float *g_idata, 
                                       int n, int baseIndex,
                                       int& ai, int& bi, 
                                       int& mem_ai, int& mem_bi, 
                                       int& bankOffsetA, int& bankOffsetB)
{
    int thid = threadIdx.x;
    mem_ai = baseIndex + threadIdx.x;
    mem_bi = mem_ai + blockDim.x;

    ai = thid;
    bi = thid + blockDim.x;

    // compute spacing to avoid bank conflicts
    bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    bankOffsetB = CONFLICT_FREE_OFFSET(bi);

    // Cache the computational window in shared memory
    // pad values beyond n with zeros
    s_data[ai + bankOffsetA] = g_idata[mem_ai]; 
    
    if (isNP2) // compile-time decision
    {
        s_data[bi + bankOffsetB] = (bi < n) ? g_idata[mem_bi] : 0; 
    }
    else
    {
        s_data[bi + bankOffsetB] = g_idata[mem_bi]; 
    }
}

template <bool isNP2>
__device__ void storeSharedChunkToMem(float* g_odata, 
                                      const float* s_data,
                                      int n, 
                                      int ai, int bi, 
                                      int mem_ai, int mem_bi,
                                      int bankOffsetA, int bankOffsetB)
{
    __syncthreads();

    // write results to global memory
    g_odata[mem_ai] = s_data[ai + bankOffsetA]; 
    if (isNP2) // compile-time decision
    {
        if (bi < n)
            g_odata[mem_bi] = s_data[bi + bankOffsetB]; 
    }
    else
    {
        g_odata[mem_bi] = s_data[bi + bankOffsetB]; 
    }
}

template <bool storeSum>
__device__ void clearLastElement(float* s_data, 
                                 float *g_blockSums, 
                                 int blockIndex)
{
    if (threadIdx.x == 0)
    {
        int index = (blockDim.x << 1) - 1;
        index += CONFLICT_FREE_OFFSET(index);
        
        if (storeSum) // compile-time decision
        {
            // write this block's total sum to the corresponding index in the blockSums array
            g_blockSums[blockIndex] = s_data[index];
        }

        // zero the last element in the scan so it will propagate back to the front
        s_data[index] = 0;
    }
}



__device__ unsigned int buildSum(float *s_data)
{
    unsigned int thid = threadIdx.x;
    unsigned int stride = 1;
    
    // build the sum in place up the tree
    for (int d = blockDim.x; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)      
        {
            int i  = __mul24(__mul24(2, stride), thid);
            int ai = i + stride - 1;
            int bi = ai + stride;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            s_data[bi] += s_data[ai];
        }

        stride *= 2;
    }

    return stride;
}

__device__ void scanRootToLeaves(float *s_data, unsigned int stride)
{
     unsigned int thid = threadIdx.x;

    // traverse down the tree building the scan in place
    for (int d = 1; d <= blockDim.x; d *= 2)
    {
        stride >>= 1;

        __syncthreads();

        if (thid < d)
        {
            int i  = __mul24(__mul24(2, stride), thid);
            int ai = i + stride - 1;
            int bi = ai + stride;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            float t  = s_data[ai];
            s_data[ai] = s_data[bi];
            s_data[bi] += t;
        }
    }
}

template <bool storeSum>
__device__ void prescanBlock(float *data, int blockIndex, float *blockSums)
{
    int stride = buildSum(data);               // build the sum in place up the tree
    clearLastElement<storeSum>(data, blockSums, 
                               (blockIndex == 0) ? blockIdx.x : blockIndex);
    scanRootToLeaves(data, stride);            // traverse down tree to build the scan 
}

template <bool storeSum, bool isNP2>
__global__ void prescan(float *g_odata, 
                        const float *g_idata, 
                        float *g_blockSums, 
                        int n, 
                        int blockIndex, 
                        int baseIndex)
{
    int ai, bi, mem_ai, mem_bi, bankOffsetA, bankOffsetB;
    extern __shared__ float s_data[];

    // load data into shared memory
    loadSharedChunkFromMem<isNP2>(s_data, g_idata, n, 
                                  (baseIndex == 0) ? 
                                  __mul24(blockIdx.x, (blockDim.x << 1)):baseIndex,
                                  ai, bi, mem_ai, mem_bi, 
                                  bankOffsetA, bankOffsetB); 
    // scan the data in each block
    prescanBlock<storeSum>(s_data, blockIndex, g_blockSums); 
    // write results to device memory
    storeSharedChunkToMem<isNP2>(g_odata, s_data, n, 
                                 ai, bi, mem_ai, mem_bi, 
                                 bankOffsetA, bankOffsetB);  
}


__global__ void uniformAdd(float *g_data, 
                           float *uniforms, 
                           int n, 
                           int blockOffset, 
                           int baseIndex)
{
    __shared__ float uni;
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];
    
    unsigned int address = __mul24(blockIdx.x, (blockDim.x << 1)) + baseIndex + threadIdx.x; 

    __syncthreads();
    
    // note two adds per thread
    g_data[address]              += uni;
    g_data[address + blockDim.x] += (threadIdx.x + blockDim.x < n) * uni;
}

inline bool 
isPowerOfTwo(int n)
{
    return ((n&(n-1))==0) ;
}

inline int 
floorPow2(int n)
{
    int exp;
    frexp((float)n, &exp);
    return 1 << (exp - 1);
}

float** g_scanBlockSums;
unsigned int g_numEltsAllocated = 0;
unsigned int g_numLevelsAllocated = 0;

void preallocBlockSums(unsigned int maxNumElements) {
    g_numEltsAllocated = maxNumElements;

    unsigned int blockSize = BLOCK_SIZE; // max size of the thread blocks
    unsigned int numElts = maxNumElements;

    int level = 0;

    do
    {       
        unsigned int numBlocks = 
            max(1, (int)ceil((float)numElts / (2.f * blockSize)));
        if (numBlocks > 1)
        {
            level++;
        }
        numElts = numBlocks;
    } while (numElts > 1);

    g_scanBlockSums = (float**) malloc(level * sizeof(float*));
    g_numLevelsAllocated = level;
    
    numElts = maxNumElements;
    level = 0;
    
    do
    {       
        unsigned int numBlocks = 
            max(1, (int)ceil((float)numElts / (2.f * blockSize)));
        if (numBlocks > 1) 
        {
            hipMalloc((void**) &g_scanBlockSums[level++], numBlocks * sizeof(float));
        }
        numElts = numBlocks;
    } while (numElts > 1);
}

void deallocBlockSums()
{
    for (int i = 0; i < g_numLevelsAllocated; i++)
    {
        hipFree(g_scanBlockSums[i]);
    }
   
    free((void**)g_scanBlockSums);

    g_scanBlockSums = 0;
    g_numEltsAllocated = 0;
    g_numLevelsAllocated = 0;
}


void prescanArrayRecursive(float *outArray, 
                           const float *inArray, 
                           int numElements, 
                           int level)
{
    unsigned int blockSize = BLOCK_SIZE; // max size of the thread blocks
    unsigned int numBlocks = 
        max(1, (int)ceil((float)numElements / (2.f * blockSize)));
    unsigned int numThreads;

    if (numBlocks > 1)
        numThreads = blockSize;
    else if (isPowerOfTwo(numElements))
        numThreads = numElements / 2;
    else
        numThreads = floorPow2(numElements);

    unsigned int numEltsPerBlock = numThreads * 2;

    // if this is a non-power-of-2 array, the last block will be non-full
    // compute the smallest power of 2 able to compute its scan.
    unsigned int numEltsLastBlock = 
        numElements - (numBlocks-1) * numEltsPerBlock;
    unsigned int numThreadsLastBlock = max(1, numEltsLastBlock / 2);
    unsigned int np2LastBlock = 0;
    unsigned int sharedMemLastBlock = 0;
    
    if (numEltsLastBlock != numEltsPerBlock)
    {
        np2LastBlock = 1;

        if(!isPowerOfTwo(numEltsLastBlock))
            numThreadsLastBlock = floorPow2(numEltsLastBlock);    
        
        unsigned int extraSpace = (2 * numThreadsLastBlock) / NUM_BANKS;
        sharedMemLastBlock = 
            sizeof(float) * (2 * numThreadsLastBlock + extraSpace);
    }

    // padding space is used to avoid shared memory bank conflicts
    unsigned int extraSpace = numEltsPerBlock / NUM_BANKS;
    unsigned int sharedMemSize = 
        sizeof(float) * (numEltsPerBlock + extraSpace);

    // setup execution parameters
    // if NP2, we process the last block separately
    dim3  grid(max(1, numBlocks - np2LastBlock), 1, 1); 
    dim3  threads(numThreads, 1, 1);

    // execute the scan
    if (numBlocks > 1)
    {
        prescan<true, false><<< grid, threads, sharedMemSize >>>(outArray, 
                                                                 inArray, 
                                                                 g_scanBlockSums[level],
                                                                 numThreads * 2, 0, 0);
        if (np2LastBlock)
        {
            prescan<true, true><<< 1, numThreadsLastBlock, sharedMemLastBlock >>>
                (outArray, inArray, g_scanBlockSums[level], numEltsLastBlock, 
                 numBlocks - 1, numElements - numEltsLastBlock);
        }

        // After scanning all the sub-blocks, we are mostly done.  But now we 
        // need to take all of the last values of the sub-blocks and scan those.  
        // This will give us a new value that must be sdded to each block to 
        // get the final results.
        // recursive (CPU) call
        prescanArrayRecursive(g_scanBlockSums[level], 
                              g_scanBlockSums[level], 
                              numBlocks, 
                              level+1);

        uniformAdd<<< grid, threads >>>(outArray, 
                                        g_scanBlockSums[level], 
                                        numElements - numEltsLastBlock, 
                                        0, 0);
        if (np2LastBlock)
        {
            uniformAdd<<< 1, numThreadsLastBlock >>>(outArray, 
                                                     g_scanBlockSums[level], 
                                                     numEltsLastBlock, 
                                                     numBlocks - 1, 
                                                     numElements - numEltsLastBlock);
        }
    }
    else if (isPowerOfTwo(numElements))
    {
        prescan<false, false><<< grid, threads, sharedMemSize >>>(outArray, inArray,
                                                                  0, numThreads * 2, 0, 0);
    }
    else
    {
         prescan<false, true><<< grid, threads, sharedMemSize >>>(outArray, inArray, 
                                                                  0, numElements, 0, 0);
    }
}

void prescanArray(float *outArray, float *inArray, int numElements)
{
    prescanArrayRecursive(outArray, inArray, numElements, 0);
}

void scan(int num_elements, float* h_data) {   
    // allocate device memory input and output arrays
    float* d_idata = NULL;
    float* d_odata = NULL;
    hipMalloc( (void**) &d_idata, sizeof(float) * num_elements);
    hipMalloc( (void**) &d_odata, sizeof(float) * num_elements);
    
    // copy host memory to device input array
    hipMemcpy( d_idata, h_data, sizeof(float) * num_elements, hipMemcpyHostToDevice);
    
    // run 
    preallocBlockSums(num_elements);
    prescanArray(d_odata, d_idata, num_elements);
    
    // copy device memory to host input array
    hipMemcpy(h_data, d_odata, sizeof(float) * num_elements, hipMemcpyDeviceToHost);
    
    deallocBlockSums();
    hipFree(d_odata);
    hipFree(d_idata);
}

#endif // _PRESCAN_CU_
