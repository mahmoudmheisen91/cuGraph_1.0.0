#include "mainPPS.hpp"

void test_scan(int size) {
	// declare host and device variable:
	int *host_in, *host_out, *dev_in, *dev_out, n = size;
    int *test_out;
	float time1;
	hipEvent_t start, stop; 
	
	dim3 grid(1);
	dim3 block(n);

	// allocate host memory:
	host_in = new int[n];
	host_out = new int[n];
    test_out = new int[n];
	
	// allocate device memory:
	hipMalloc((void**)&dev_in, n * sizeof(int));
	hipMalloc((void**)&dev_out, n * sizeof(int));
	
	// fill 
	for(int i = 0; i < n; i++) {
		host_in[i] = i+1;
        test_out[i] = i+1;
	}
	
	// copy host vars to device vars:
	hipMemcpy(dev_in, host_in, n*sizeof(int), hipMemcpyHostToDevice);

    // start Device kernal:
	hipEventCreate(&start); 
	hipEventCreate(&stop);
	hipEventRecord(start, 0); 

	cuScan<<<grid, block, n * sizeof(float)>>>(dev_out, dev_in, n);

	hipEventRecord(stop, 0); 
	hipEventSynchronize(stop);  
	hipEventElapsedTime(&time1, start, stop); 
	hipEventDestroy(start); 
	hipEventDestroy(stop);
	
	// print time:
	cout << endl << "Kernal time = " << time1 << " ms, size = " << n << endl;
	
	// copy device vars to host vars:
	hipMemcpy(host_out, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
	
    // check outout:
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    exclusive_scan_sum(test_out, size);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time1, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // print time:
    cout << endl << "Host time = " << time1 << " ms, size = " << n << endl;

	for(int i = 0; i < n; i++) {
        if(test_out[i] != host_out[i])
            cout << test_out[i] << " " << host_out[i] << endl;
	}

	// free host memory:
	delete host_in;
	delete host_out;
    delete test_out;
	
	// free device memory:
	hipFree(dev_in);
	hipFree(dev_out);
}
