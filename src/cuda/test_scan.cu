#include "mainPPS.hpp"

void test_scan(int size, int num) {
	// declare host and device variable:
	int *host_in, *host_out, *dev_in, *dev_out, n = size;
	float time1;
	hipEvent_t start, stop; 
	
	dim3 grid(1);
	dim3 block(n);

	// allocate host memory:
	host_in = new int[n];
	host_out = new int[n];
	
	// allocate device memory:
	hipMalloc((void**)&dev_in, n * sizeof(int));
	hipMalloc((void**)&dev_out, n * sizeof(int));
	
	// fill 
	for(int i = 0; i < n; i++) {
		host_in[i] = i+1;
	}
	
	// copy host vars to device vars:
	hipMemcpy(dev_in, host_in, n*sizeof(int), hipMemcpyHostToDevice);
int shared = n*sizeof(int);
	// start Device kernal:
	hipEventCreate(&start); 
	hipEventCreate(&stop);
	hipEventRecord(start, 0); 
	cuScan<<<grid, block>>>(dev_out, dev_in, n);
	hipEventRecord(stop, 0); 
	hipEventSynchronize(stop);  
	hipEventElapsedTime(&time1, start, stop); 
	hipEventDestroy(start); 
	hipEventDestroy(stop);
	
	// print time:
	cout << endl << "Kernal time = " << time1 << " ms, size = " << n << endl;
	
	// copy device vars to host vars:
	hipMemcpy(host_out, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
	
	for(int i = 0; i < n; i++) {
		cout << host_out[i] << " ";
	}
	
	cout << endl;
	// free host memory:
	delete host_in;
	delete host_out;
	
	// free device memory:
	hipFree(dev_in);
	hipFree(dev_out);
}
