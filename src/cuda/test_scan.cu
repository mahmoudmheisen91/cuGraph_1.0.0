#include "mainPPS.hpp"

void test_scan(int size) {
	// declare host and device variable:
	int *host_in, *host_out, *dev_in, *dev_out, n = size;
    int *test_out, *mid1, *mid1_out, *mid2, *mid2_out;
	float time1;
	hipEvent_t start, stop; 

	// allocate host memory:
	host_in = new int[n];
	mid1 = new int[4];
	mid1_out = new int[4];
	host_out = new int[n];
    test_out = new int[n];
	
	// allocate device memory:
	hipMalloc((void**)&dev_in, n * sizeof(int));
	hipMalloc((void**)&dev_out, n * sizeof(int));
	hipMalloc((void**)&mid2_out, 4 * sizeof(int));
	hipMalloc((void**)&mid2, 4 * sizeof(int));
	
	// fill 
	for(int i = 0; i < n; i++) {
		host_in[i] = i+1;
        test_out[i] = i+1;
	}
	
	// copy host vars to device vars:
	hipMemcpy(dev_in, host_in, n*sizeof(int), hipMemcpyHostToDevice);

    // start Device kernal:
	hipEventCreate(&start); 
	hipEventCreate(&stop);
	hipEventRecord(start, 0); 

	scanGlobal<<<4, 256>>>(dev_out, dev_in, 256);
	hipMemcpy(host_out, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
	/*mid1[0] = host_out[255];
	mid1[1] = host_out[511];
	mid1[2] = host_out[511+256];
	mid1[3] = host_out[1023];
	hipMemcpy(mid2, mid1, 4*sizeof(int), hipMemcpyHostToDevice);
	scanGlobal<<<1, 4>>>(mid2_out, mid2, 4);
	hipMemcpy(mid1_out, mid2_out, 4 * sizeof(int), hipMemcpyDeviceToHost);
	
	for(int i = 256; i < 512; i++) {
       //host_out[i] += mid1_out[0];
	}
	
	for(int i = 512; i < 512 + 256; i++) {
       host_out[i] += mid1_out[1];
	}
	
	for(int i = 512 + 256; i < 1024; i++) {
       host_out[i] += mid1_out[2];
	}*/

	hipEventRecord(stop, 0); 
	hipEventSynchronize(stop);  
	hipEventElapsedTime(&time1, start, stop); 
	hipEventDestroy(start); 
	hipEventDestroy(stop);
	
	// print time:
	cout << endl << "Kernal time = " << time1 << " ms, size = " << n << endl;
	
	// copy device vars to host vars:
	//hipMemcpy(host_out, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
	
    exclusive_scan_sum(test_out, size);

	for(int i = 0; i < 512; i++) {
        //if(test_out[i] != host_out[i])
            cout << host_out[i] << " " ;
	}

	// free host memory:
	delete host_in;
	delete host_out;
    delete test_out;
	
	// free device memory:
	hipFree(dev_in);
	hipFree(dev_out);
}
