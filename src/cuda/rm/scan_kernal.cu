#include "hip/hip_runtime.h"
#include "mainPPS.hpp"

__global__ void scanGlobal(int *g_odata, int *g_idata, int n) {

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	cuScanBlock(g_odata, g_idata, n, id);
}

__device__ void cuScanBlock(int *g_odata, int *g_idata, int n) {
	//extern __shared__ float temp[];
	
	int thid = threadIdx.x;
	int offset = 1;
	int t=0;
	//temp[thid] = g_idata[thid];
	
	// build sum in place up the tree:
	for (int d = n>>1; d > 0; d >>= 1) { 
		__syncthreads();
		if (thid < d) {
			int ai = offset*(2*thid+1)-1;  
			int bi = offset*(2*thid+2)-1;  
			g_idata[bi] += g_idata[ai];
		}
		offset *= 2;
	}
	
	// clear the last element:
	if (thid==0) {
		sum[t++] = g_idata[n - 1];
		g_idata[n - 1] = 0; 
	}
	
	// traverse down tree & build scan:
	for (int d = 1; d < n; d *= 2) { 
		offset >>= 1;
		__syncthreads();
		if (thid < d) {
			int ai = offset*(2*thid+1)-1;  
			int bi = offset*(2*thid+2)-1;  
			int t = g_idata[ai];
			g_idata[ai] = g_idata[bi];
			g_idata[bi] += t;
		}
	}
	__syncthreads();
	g_odata[thid] = g_idata[thid];
} 



