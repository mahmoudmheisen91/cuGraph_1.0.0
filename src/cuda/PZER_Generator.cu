/*
 * PZER_Generator.cu
 *
 *  Created: 2015-05-10, Modified: 2015-07-26
 *
 */

// Must be defined before any header to use GPU_TIMER:
// Must be defined in just one file of the project (to be fixed):
//#define TIMER

// Headers includes:
#include <cuda/kernels.cuh>
#include <cuda/Cuda_Prototypes_Macros.h>

// Parallel CUDA random graph generator 2, PZER:
void PZER_Generator(bool *content,   		/* in\out */ 
				   	float skipping_prob,	/* in */ 
				   	int lambda, 			/* in */ 
				   	int vertex_num, 		/* in */ 
				   	int edges_num)			/* in */ 
{
    // Const:
    const unsigned int B = GRID_SIZE * BLOCK_SIZE;
    const unsigned int seed = time(0) - 1000000000;
    
    // Declerations:
    bool *d_content; float *d_R;
    int *d_L, *d_S, *d_block_results, L;   

    // Allocations:
    CUDA_SAFE_CALL(hipMalloc((void**) &d_content, vertex_num * vertex_num * sizeof(bool)));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_R, B * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_S, B * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_L, sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_block_results, GRID_SIZE * sizeof(int)));

    // Init d_content with false values using hipMemset:
    CUDA_SAFE_CALL(hipMemset(d_content, false, vertex_num * vertex_num * sizeof(bool)));

    // Main loop:
    L = 0;
    while(L < edges_num) {
        random_number_generator_kernel 	<<<8		 , BLOCK_SIZE>>> (seed, B, d_R);						// Generate B rand numbers
        skipValue_kernel				<<<GRID_SIZE , BLOCK_SIZE>>> (d_R, B, skipping_prob, d_S);			// predicte skiped edges
		global_scan_kernel_1 			<<<GRID_SIZE , BLOCK_SIZE>>> (d_S, d_block_results);				// scan k1
		global_scan_kernel_2 			<<<1		 , BLOCK_SIZE>>> (d_block_results);						// scan k2
		global_scan_kernel_3 			<<<GRID_SIZE , BLOCK_SIZE>>> (d_S, d_block_results);				// scan k3
        update_cancatate_kernel			<<<GRID_SIZE , BLOCK_SIZE>>> (d_S, B, L);							// concanate scaned values
        addEdges_kernel					<<<GRID_SIZE , BLOCK_SIZE>>> (d_S, B, vertex_num, d_content, d_L);	// add edge to the graph
		
		// Update the value of L:
        CUDA_SAFE_CALL(hipMemcpy(&L, d_L, sizeof(int), hipMemcpyDeviceToHost));	
    }

	// Copy content from Device To Host:
    CUDA_SAFE_CALL(hipMemcpy(content, d_content, sizeof(bool) * vertex_num * vertex_num, hipMemcpyDeviceToHost));

    // Free the device:
    CUDA_SAFE_CALL(hipFree(d_S));
    CUDA_SAFE_CALL(hipFree(d_R));
    CUDA_SAFE_CALL(hipFree(d_L));
    CUDA_SAFE_CALL(hipFree(d_content));
    CUDA_SAFE_CALL(hipFree(d_block_results));
    CUDA_SAFE_CALL(hipDeviceReset());
}





