#include "mainPPS.hpp"

void test_scan(int size, int num) {
	// declare host and device variable:
	int *host_masterSeed, *host_size, *dev_masterSeed, *dev_size;
	float *host_PRNG, *dev_PRNG, time1;
	hipEvent_t start, stop; 
	
	dim3 grid(gridSize-1);
	dim3 block(blockSize);

	// allocate host memory:
	host_masterSeed = new int[1];
	host_size = new int[1];
	host_PRNG = new float[size];
	
	// allocate device memory:
	hipMalloc((void**)&dev_masterSeed, sizeof(int));
	hipMalloc((void**)&dev_size, sizeof(int));
	hipMalloc((void**)&dev_PRNG, size * sizeof(float));
	
	// fill 
	host_masterSeed[0] = time(0) - 1000000000; // between 1 and m-1
	host_size[0] = size;
	
	// copy host vars to device vars:
	hipMemcpy(dev_masterSeed, host_masterSeed, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_size, host_size, sizeof(int), hipMemcpyHostToDevice);

	// start Device kernal:
	hipEventCreate(&start); 
	hipEventCreate(&stop);
	hipEventRecord(start, 0); 
	random_number_generator_kernal<<<grid, block>>>(dev_masterSeed, dev_size, dev_PRNG);
	hipEventRecord(stop, 0); 
	hipEventSynchronize(stop);  
	hipEventElapsedTime(&time1, start, stop); 
	hipEventDestroy(start); 
	hipEventDestroy(stop);
	
	// print time:
	cout << endl << "Kernal time = " << time1 << " ms, size = " << size << endl;
	
	// copy device vars to host vars:
	hipMemcpy(host_PRNG, dev_PRNG, size * sizeof(float), hipMemcpyDeviceToHost);
	
	// free host memory:
	delete host_masterSeed;
	delete host_size;
	delete host_PRNG;
	
	// free device memory:
	hipFree(dev_masterSeed);
	hipFree(dev_size);
	hipFree(dev_PRNG);
}
