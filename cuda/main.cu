#include "hip/hip_runtime.h"
#include "main.hpp"

int main(int argc, char **argv) {

	// declare host and device variable:
	int *host_masterSeed, *host_itemsPerThread;
	float *host_PRNG;
	int *dev_masterSeed, *dev_itemsPerThread;
	float *PRNG, *dev_PRNG;
	
	// allocate host memory:
	int a = 102400;
	host_masterSeed = new int[1];
	host_itemsPerThread = new int[1];
	host_PRNG = new float[a*N];
	PRNG = new float[a*N];
	
	// allocate device memory:
	hipMalloc((void**)&dev_masterSeed, sizeof(int));
	hipMalloc((void**)&dev_itemsPerThread, sizeof(int));
	hipMalloc((void**)&dev_PRNG, a * N * sizeof(float));
	
	// fill 
	host_masterSeed[0] = time(0);
	host_itemsPerThread[0] = a;
	
	// host kernal:
	//random_number_generator_host(PRNG);
	
	// copy host vars to device vars:
	hipMemcpy(dev_masterSeed, host_masterSeed, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_itemsPerThread, host_itemsPerThread, sizeof(int), hipMemcpyHostToDevice);
	
	// start Device kernal:
	random_number_generator_kernal<<<gridSize, blockSize>>>(dev_masterSeed, dev_itemsPerThread, dev_PRNG);
	
	// start Host Kernal:
	random_number_generator_host(host_masterSeed, host_itemsPerThread, PRNG);
	
	// copy device vars to host vars:
	hipMemcpy(host_PRNG, dev_PRNG, a * N * sizeof(float), hipMemcpyDeviceToHost);
	
	// print output:
	int num = 2;
	int result[num];
	fill(result, result+num, 0);
	
	for(int i = 0; i < a * N; i++) {
		result[(int)(host_PRNG[i]*num)]++;
	}
	
	cout << "avg = " << a * N / num << endl;
	for(int i = 0; i < num; i++) {
		cout << result[i] << " ";
	}
	
	fill(result, result+num, 0);
	
	for(int i = 0; i < a * N; i++) {
		result[(int)(PRNG[i]*num)]++;
	}
	
	cout <<endl;
	for(int i = 0; i < num; i++) {
		cout << result[i] << " ";
	}
	
	// free host memory:
	delete host_masterSeed;
	delete host_itemsPerThread;
	delete host_PRNG;
	delete PRNG;
	
	// free device memory:
	hipFree(dev_masterSeed);
	hipFree(dev_itemsPerThread);
	hipFree(dev_PRNG);
	
    return 0;
}

__global__ void random_number_generator_kernal(int *masterSeed, int *itemsPerThread, float *PRNG) {
	long int a = 16807;                      // same as apple c++ imp
	long int m = 2147483647;                 // 2^31 − 1
	float rec  = 1.0 / m;
	
	long int seed = *masterSeed + threadIdx.x;
	
	long int theta;
	long int temp;
	int to = *itemsPerThread;
	for (int i = 0; i < to; i++) {
		temp = seed * a;                       // seed = Xn , c = 0
		theta = temp - m * floor(temp * rec);  // is the same as (temp mod m) ((Xn * a) mod m)
		seed = theta;
		PRNG[i + to * threadIdx.x] = (float)theta/m;
	}
}







