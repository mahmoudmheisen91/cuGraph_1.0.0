#include "hip/hip_runtime.h"
#include "main.hpp"

__global__ void random_number_generator_kernal(int *masterSeed, int *size, float *PRNG) {
	long int a = 16807;                      // same as apple c++ imp
	long int m = 2147483647;                 // 2^31 − 1
	float rec  = 1.0 / m;
	
	//int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int tid = (blockIdx.y * gridDim.x + blockIdx.x)*(blockDim.y * blockDim.x) + threadIdx.y * blockDim.x + threadIdx.x; 
	
	//int tid = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x)*blockDim.x); 
	long int seed = *masterSeed + tid;
	
	long int theta, temp;
	if (tid < *size) {
		temp = seed * a;                       // seed = Xn , c = 0
		theta = temp - m * floor(temp * rec);  // is the same as (temp mod m) ((Xn * a) mod m)
		//seed = theta;
		PRNG[tid] = (float)theta/m;
	}
}
