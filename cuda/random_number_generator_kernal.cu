#include "hip/hip_runtime.h"
#include "main.hpp"

__global__ void random_number_generator_kernal(int *masterSeed, int *size, float *PRNG) {
	long int a = 16807;                      // same as apple c++ imp
	long int m = 2147483647;                 // 2^31 − 1
	float rec  = 1.0 / m;
	
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//int y = threadIdx.y + blockIdx.y * blockDim.y;
	//int tid = x + y * blockDim.x * gridDim.x;

	long int seed = *masterSeed + tid;
	
	long int theta, temp;
	while (tid < *size) {
		temp = seed * a;                       // seed = Xn , c = 0
		theta = temp - m * floor(temp * rec);  // is the same as (temp mod m) ((Xn * a) mod m)
		PRNG[tid] = (float)theta/m;			   // between 1/m - 1
		tid += blockDim.x * gridDim.x;
	}
}
